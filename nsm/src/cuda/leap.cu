#include "hip/hip_runtime.h"
#include "../../include/cuda/leap.cuh"

__device__ bool is_critical(int * state, int * reactants, int * products, int sbi, int ri)
{
	bool crit = false;
	for (int i = 0; i < SPC; i++) {
		crit = crit || ((reactants[GET_COEFF(i, ri)] - products[GET_COEFF(i, ri)]) * NC > state[GET_SPI(i, sbi)]);
	}

	return crit;
}

__device__ float compute_g(int * state, int * reactants, int sbi, int spi)
{

	int hor = HOR(reactants, spi);

#ifdef DEBUG
	if (hor < 1 || hor > 2) {
		printf(">>>>> ARGH! @ compute_g(%d, %d): hor = %d\n", sbi, spi, hor);
	}
#endif

	int x = 0;
	switch (hor) {
	case 1:
		return 1;
	case 2:
		return 2;
	case 3:
		x = state[GET_SPI(spi, sbi)];
		if (x == 1) { // TODO: is 1.0 / +Inf == 0? can we use this to avoid the check?
			return 2.0;
		}
		return 2.0 + 1.0 / (x - 1);
	default:
		return 0; // nope
	}
}

__device__ int HOR(int * reactants, int spi)
{
	int max_hor = 0;
	bool is_bi_reaction = false;

	for (int i = 0; i < RC; i++) {

		// if spi is not a reactant of the current
		// reaction, continue with the next one.
		if (reactants[GET_COEFF(spi, i)] == 0) {
			continue;
		}

		// sum all the coeff. of the current
		// reaction to compute its order.
		int hor = 0;
		for (int j = 0; j < SPC; j++) {
			int c = reactants[GET_COEFF(j, i)];
			hor += c;
			// check if ri requires 2 molecules of spi
			if (j == spi && c == 2) {
				is_bi_reaction = true; // TODO: replace with branchless code
			}
		}

		max_hor = max(hor, max_hor);
	}

	if (is_bi_reaction) {
		max_hor = 3;
	}

	return max_hor;

}

__device__ float compute_mu(int * state, int * reactants, int * products, int sbi, int spi, float * react_rates_array)
{
	float mu = 0.0;

	for (int i = 0; i < RC; i++) {

		// when computing mu we only sum over non-critical reactions
		if(is_critical(state, reactants, products, sbi, i)) {
			continue;
		}

		// mu is the sum of (change_vector) * (reaction_rate) over
		// non-critical reactions.
		mu += (products[GET_SPI(spi, sbi)] - reactants[GET_SPI(spi, sbi)]) * react_rates_array[GET_RR(i, sbi)];
	}

	return mu;
}

__device__ float compute_sigma2(int * state, int * reactants, int * products, int sbi, int spi, float * react_rates_array)
{
	float sigma2 = 0.0;

	for (int i = 0; i < RC; i++) {

		// when computing sigma2 we only sum over non-critical reactions
		if(is_critical(state, reactants, products, sbi, i)) {
			continue;
		}

		// sigma2 is the sum of (change_vector)^2 * (reaction_rate) over
		// non-critical reactions.
		int v = products[GET_SPI(spi, sbi)] - reactants[GET_SPI(spi, sbi)];
		sigma2 += (v*v) * react_rates_array[GET_RR(i, sbi)];
	}

	return sigma2;
}


