#include "hip/hip_runtime.h"
#include "../../include/cuda/rates.cuh"

__device__ float react_rate(int * state, int * reactants, int sbc, int spc, int rc, int sbi, int ri, float * rrc)
{
	// search for the first specie in the reactions array that
	// does have a positive coefficent
	int index1 = ri;
	int specie_index = 0;
	while (reactants[index1] == 0) {
		index1 += rc;
		specie_index++;
	}

	if (reactants[index1] == 2) {    // bi_same reaction type
		// get specie count for that specie in the current subvolume
		// int specie_count = state[specie_index * subvolumes_count + subvolume_index];
		int specie_count = state[GET_SPI(specie_index, sbi, sbc)];
		return 0.5 * specie_count * (specie_count - 1) * rrc[ri];
	}

	// if specie_index == # of species we are in a uni reaction
	if (specie_index != spc - 1) {

		// search for a possibile other specie with positive coefficient
		int index2 = index1 + rc;
		int specie_index2 = specie_index + 1;
		while (reactants[index2] == 0 && index2 < (spc * rc - 1)) {
			index2 += rc;
			specie_index2++;
		}

		if (reactants[index2] != 0) {    // bi_diff reaction type
			int specie1_count = state[GET_SPI(specie_index, sbi, sbc)];
			int specie2_count = state[GET_SPI(specie_index2, sbi, sbc)];
			return specie1_count * specie2_count * rrc[ri];
		}
	}

	// uni reaction type
	int specie_count = state[GET_SPI(specie_index, sbi, sbc)];
	return specie_count * rrc[ri];
}

__device__ void react_rates(int * state, int * reactants, int sbc, int spc, int rc, float * rrc,
		float * react_rates_array)
{
	int sbi = blockIdx.x * blockDim.x + threadIdx.x;
	if (sbi >= sbc)
		return;

	for (int i = 0; i < rc; i++) {
		react_rates_array[sbc * i + sbi] = react_rate(state, reactants, sbc, spc, rc, sbi, i, rrc);
	}
}

__device__ void diff_rates(int * state, int sbc, int spc, float * drc, float * diff_rates_array)
{
	int sbi = blockIdx.x * blockDim.x + threadIdx.x;
	if (sbi >= sbc)
		return;

	for (int i = 0; i < spc; i++) {
		diff_rates_array[sbc * i + sbi] = drc[i] * state[GET_SPI(i, sbi, sbc)];
	}
}

__device__ void update_rate_matrix(int * topology, int sbc, int spc, int rc, float * rate_matrix,
		float * react_rates_array, float * diff_rates_array)
{
	int sbi = blockIdx.x * blockDim.x + threadIdx.x;
	if (sbi >= sbc)
		return;

	// sum reaction rates
	float react_sum = 0.0;
	for (int i = 0; i < rc; i++)
		react_sum += react_rates_array[sbc * i + sbi];

	// sum diffusion rates
	float diff_sum = 0.0;
	for (int i = 0; i < spc; i++)
		diff_sum += diff_rates_array[sbc * i + sbi];

	// count subvolume neighbours (since diff_rate = #neighbours x diff_sum)
	int neigh_count = 0;
	for (int i = 0; i < 6; i++)
		neigh_count += (topology[sbi * 6 + i] != -1);

	diff_sum *= neigh_count;

	// write data into rate matrix
	rate_matrix[sbc * 0 + sbi] = react_sum;
	rate_matrix[sbc * 1 + sbi] = diff_sum;
	rate_matrix[sbc * 2 + sbi] = react_sum + diff_sum;
}

__global__ void compute_rates(int * state, int * reactants, int * topology, int sbc, int spc, int rc,
		float * rate_matrix, float * rrc, float * drc, float * react_rates_array, float * diff_rates_array)
{
	react_rates(state, reactants, sbc, spc, rc, rrc, react_rates_array);
	diff_rates(state, sbc, spc, drc, diff_rates_array);
	update_rate_matrix(topology, sbc, spc, rc, rate_matrix, react_rates_array, diff_rates_array);
}

void h_compute_rates(int * state, int * reactants, int * topology, int sbc, int spc, int rc, float * rate_matrix,
		float * rrc, float * drc, float * react_rates_array, float * diff_rates_array)
{
	compute_rates<<<1, sbc>>>(state, reactants, topology, sbc, spc, rc, rate_matrix, rrc, drc, react_rates_array,
			diff_rates_array);
}
