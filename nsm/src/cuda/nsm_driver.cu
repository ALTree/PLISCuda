#include <hip/hip_runtime.h>

// TODO: move includes into header
#include <thrust/device_vector.h>
#include <hiprand/hiprand_kernel.h>

#include "../../include/cuda/nsm_driver.cuh"

#include "../../include/cuda/cuda_utils.cuh"
#include "../../include/cuda/rates.cuh"
#include "../../include/cuda/nsm.cuh"
#include "../../include/cuda/constants.cuh"

__constant__ unsigned int SBC;
__constant__ int SPC;
__constant__ int RC;
__constant__ int NC;
__constant__ float EPSILON;

// #define DEBUG

namespace NSMCuda {

// TODO: rename
// we keep nsm_step as a kernel, but this will be
// the main driver and "nsm" is not the right name.
void nsm(Topology t, State s, Reactions r, float * h_rrc, float * h_drc)
{
	unsigned int sbc = t.getN();
	int spc = s.getS();
	int rc = r.getR();

	int nc = 10;
	float epsilon = 0.05;

	hipMemcpyToSymbol(HIP_SYMBOL(SBC), &sbc, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(SPC), &spc, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(RC), &rc, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(NC), &nc, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(EPSILON), &epsilon, sizeof(float));

	std::cout << "----- Allocating GPU memory ...";

	// ----- allocate and memcpy state array -----
	int * h_state = s.getState();

	int * d_state;
	gpuErrchk(hipMalloc(&d_state, sbc * spc * sizeof(int)));
	gpuErrchk(hipMemcpy(d_state, h_state, sbc * spc * sizeof(int), hipMemcpyHostToDevice));

	// ----- allocate and memcpy reactants and products arrays -----
	int * h_reactants = r.getReactants();
	int * h_products = r.getProducts();

	int * d_reactants;
	int * d_products;
	gpuErrchk(hipMalloc(&d_reactants, spc * rc * sizeof(int)));
	gpuErrchk(hipMalloc(&d_products, spc * rc * sizeof(int)));
	gpuErrchk(hipMemcpy(d_reactants, h_reactants, spc * rc * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_products, h_products, spc * rc * sizeof(int), hipMemcpyHostToDevice));

	// ----- allocate and memcpy topology array -----
	int * h_topology = t.getNeighboursArray();

	int * d_topology;
	gpuErrchk(hipMalloc(&d_topology, 6 * sbc * sizeof(int)));
	gpuErrchk(hipMemcpy(d_topology, h_topology, 6 * sbc * sizeof(int), hipMemcpyHostToDevice));

	// ----- allocate rate matrix -----
	float * d_rate_matrix;
	gpuErrchk(hipMalloc(&d_rate_matrix, 3 * sbc * sizeof(float)));

	// ----- allocate and memcpy rrc and drc -----
	float * d_rrc;
	float * d_drc;
	gpuErrchk(hipMalloc(&d_rrc, rc * sizeof(float)));
	gpuErrchk(hipMalloc(&d_drc, spc * sizeof(float)));
	gpuErrchk(hipMemcpy(d_rrc, h_rrc, rc * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_drc, h_drc, spc * sizeof(float), hipMemcpyHostToDevice));

	// ----- allocate react_rates and diff_rates array
	float * d_react_rates_array;
	float * d_diff_rates_array;
	gpuErrchk(hipMalloc(&d_react_rates_array, sbc * rc * sizeof(float)));
	gpuErrchk(hipMalloc(&d_diff_rates_array, sbc * spc * sizeof(float)));

	// ----- allocate next_event thrust  vector
	thrust::device_vector<float> tau(sbc);

	// ----- allocate and initialize prng array
	hiprandStateMRG32k3a* d_prngstate;
	gpuErrchk(hipMalloc(&d_prngstate, sbc * sizeof(hiprandStateMRG32k3a)));
	fill_prngstate_array<<<1, sbc>>>(d_prngstate);

	// zero GPU memory, just to be sure
	// TODO: remove(?)
	gpuErrchk(hipMemset(d_rate_matrix, 0, 3 * sbc * sizeof(float)));
	gpuErrchk(hipMemset(d_react_rates_array, 0, sbc * rc * sizeof(float)));
	gpuErrchk(hipMemset(d_diff_rates_array, 0, sbc * spc * sizeof(float)));

	std::cout << " done!\n";

	std::cout << "--- Starting nsm \n";

	std::cout << "----- Initializing rate matrix... ";

	compute_rates<<<1, sbc>>>(d_state, d_reactants, d_topology, d_rate_matrix, d_rrc, d_drc, d_react_rates_array,
			d_diff_rates_array);

	std::cout << "done!\n";

	std::cout << "----- Fill initial next_event array... ";

	fill_tau_array<<<1, sbc>>>(thrust::raw_pointer_cast(tau.data()), d_rate_matrix);

	std::cout << "done!\n";

	std::cout << "----- Starting nsm iterations... \n";

	int steps = 10000;

	for (int step = 0; step < steps; step++) {

#ifdef DEBUG
		std::cout << "\n---------- step " << step << " ----------\n";

		// print state
		gpuErrchk(hipMemcpy(h_state, d_state, sbc * spc * sizeof(int), hipMemcpyDeviceToHost));
		std::cout << "-- state\n";
		for (int i = 0; i < sbc; i++) {
			std::cout << "sub " << i << ": ";
			for (int j = 0; j < spc; j++)
			std::cout << h_state[j * sbc + i] << " ";
			std::cout << "\n";
		}

		std::cout << "\n";

		// print rate matrix
		float * h_rate_matrix = new float[3 * sbc];
		gpuErrchk(hipMemcpy(h_rate_matrix, d_rate_matrix, 3 * sbc * sizeof(float), hipMemcpyDeviceToHost));
		std::cout << "-- rate matrix\n";
		for (int i = 0; i < sbc; i++) {
			std::cout << "sub " << i << ": ";
			std::cout << h_rate_matrix[i] << " ";
			std::cout << h_rate_matrix[i + sbc] << " ";
			std::cout << h_rate_matrix[i + sbc * 2] << " ";
			std::cout << "\n";
		}

		std::cout << "\n";
#endif

		int next = h_get_min_tau(tau);

		nsm_step<<<1, sbc>>>(d_state, d_reactants, d_products, d_topology, d_rate_matrix, d_rrc, d_drc,
				d_react_rates_array, d_diff_rates_array, thrust::raw_pointer_cast(tau.data()), next, d_prngstate);
	}
	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipMemcpy(h_state, d_state, sbc * spc * sizeof(int), hipMemcpyDeviceToHost));
	std::cout << "-- state\n";
	for (int i = 0; i < sbc; i++) {
		std::cout << "sub " << i << ": ";
		for (int j = 0; j < spc; j++)
			std::cout << h_state[j * sbc + i] << " ";
		std::cout << "\n";
	}

}

}
