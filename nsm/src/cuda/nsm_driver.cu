#include "hip/hip_runtime.h"
#include "../../include/cuda/nsm_driver.cuh"

__constant__ unsigned int SBC;
__constant__ int SPC;
__constant__ int RC;
__constant__ int NC;
__constant__ float EPSILON;
__constant__ int * REACTANTS;
float TAU;

namespace NSMCuda {

// TODO: rename
// we keep nsm_step as a kernel, but this will be
// the main driver and "nsm" is not the right name.
void nsm(Topology t, State s, Reactions r, float * h_rrc, float * h_drc, int steps)
{
	unsigned int sbc = t.getN();
	int spc = s.getS();
	int rc = r.getR();

	int nc = 10;    // critical event threshold
	float epsilon = 0.05;

#if LOG
	std::cout << "\n   ***   Start simulation log   ***   \n\n";
#endif

	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(SBC), &sbc, sizeof(unsigned int)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(SPC), &spc, sizeof(int)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(RC), &rc, sizeof(int)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(NC), &nc, sizeof(int)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(EPSILON), &epsilon, sizeof(float)));

#if LOG
	std::cout << "--- Allocating GPU memory... ";
#endif

	// ----- allocate and memcpy state array -----
	int * h_state = s.getState();

	int * d_state;
	gpuErrchk(hipMalloc(&d_state, sbc * spc * sizeof(int)));
	gpuErrchk(hipMemcpy(d_state, h_state, sbc * spc * sizeof(int), hipMemcpyHostToDevice));

	// ----- allocate and memcpy reactants and products arrays -----
	int * h_reactants = r.getReactants();
	int * h_products = r.getProducts();

	int * d_reactants;
	int * d_products;
	gpuErrchk(hipMalloc(&d_reactants, spc * rc * sizeof(int)));
	gpuErrchk(hipMalloc(&d_products, spc * rc * sizeof(int)));
	gpuErrchk(hipMemcpy(d_reactants, h_reactants, spc * rc * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_products, h_products, spc * rc * sizeof(int), hipMemcpyHostToDevice));

	// ----- allocate and memcpy topology array -----
	unsigned int * h_topology = t.getNeighboursArray();

	unsigned int * d_topology;
	gpuErrchk(hipMalloc(&d_topology, 6 * sbc * sizeof(unsigned int)));
	gpuErrchk(hipMemcpy(d_topology, h_topology, 6 * sbc * sizeof(unsigned int), hipMemcpyHostToDevice));

	// ----- allocate rate matrix -----
	float * d_rate_matrix;
	gpuErrchk(hipMalloc(&d_rate_matrix, 3 * sbc * sizeof(float)));

	// ----- allocate and memcpy rrc and drc -----
	float * d_rrc;
	float * d_drc;
	gpuErrchk(hipMalloc(&d_rrc, rc * sizeof(float)));
	gpuErrchk(hipMalloc(&d_drc, spc * sizeof(float)));
	gpuErrchk(hipMemcpy(d_rrc, h_rrc, rc * sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_drc, h_drc, spc * sizeof(float), hipMemcpyHostToDevice));

	// ----- allocate react_rates and diff_rates array
	float * d_react_rates_array;
	float * d_diff_rates_array;
	gpuErrchk(hipMalloc(&d_react_rates_array, sbc * rc * sizeof(float)));
	gpuErrchk(hipMalloc(&d_diff_rates_array, sbc * spc * sizeof(float)));

	// ----- allocate tau thrust vector and current_time
	thrust::device_vector<float> tau(sbc);
	float * d_current_time;
	float h_current_time = 0.0;
	gpuErrchk(hipMalloc(&d_current_time, sizeof(float)));

	// ----- allocate and initialize prng array
	hiprandStateMRG32k3a * d_prngstate;
	gpuErrchk(hipMalloc(&d_prngstate, sbc * sizeof(hiprandStateMRG32k3a)));
	fill_prngstate_array<<<1, sbc>>>(d_prngstate);

	// ----- allocate leap and cr arrays
	bool * d_leap;
	bool * d_cr;
	gpuErrchk(hipMalloc(&d_leap, sbc * sizeof(bool)));
	gpuErrchk(hipMalloc(&d_cr, sbc * sizeof(bool)));

	// zero GPU memory, just to be sure
	// TODO: remove(?)
	gpuErrchk(hipMemset(d_rate_matrix, 0, 3 * sbc * sizeof(float)));
	gpuErrchk(hipMemset(d_react_rates_array, 0, sbc * rc * sizeof(float)));
	gpuErrchk(hipMemset(d_diff_rates_array, 0, sbc * spc * sizeof(float)));
	gpuErrchk(hipMemset(d_leap, 0, sbc * sizeof(bool)));

#if LOG
	std::cout << "done!\n";
	std::cout << "--- Initializing rate matrix... ";
#endif

	compute_rates<<<1, sbc>>>(d_state, d_reactants, d_topology, d_rate_matrix, d_rrc, d_drc, d_react_rates_array,
			d_diff_rates_array);

#if LOG
	std::cout << "done!\n";
#endif

	float * h_rate_matrix;

#if LOG
	h_rate_matrix = new float[3 * sbc];
	gpuErrchk(hipMemcpy(h_rate_matrix, d_rate_matrix, 3 * sbc * sizeof(float), hipMemcpyDeviceToHost));
	print_rate_matrix(h_rate_matrix, sbc);
#endif

#if LOG
	std::cout << "--- Fill initial next_event array... ";
#endif

	fill_tau_array_leap<<<1, sbc>>>(d_state, d_reactants, d_products, d_topology, d_rate_matrix, d_react_rates_array,
			d_diff_rates_array, thrust::raw_pointer_cast(tau.data()), d_leap, d_cr, d_prngstate);

#if LOG
	// print tau array
	print_tau(tau, sbc);

	bool * h_leap = new bool[sbc];
	bool * h_cr = new bool[sbc];
	gpuErrchk(hipMemcpy(h_leap, d_leap, sbc * sizeof(bool), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_cr, d_cr, sbc * sizeof(bool), hipMemcpyDeviceToHost));
	for (int i = 0; i < sbc; i++) {
		std::cout << "sbi " << i << "] " << "leap: " << (h_leap[i] ? "yes" : "no") << ", cr: "
				<< (h_cr[i] ? "yes" : "no") << "\n";
	}
#endif

#if LOG
	std::cout << "done!\n";
#endif

#if LOG
	print_tau(tau, sbc);
	std::cout << "--- Start simulation.\n\n";
#endif

	for (int step = 1; step <= steps; step++) {

		int min_tau_sbi = h_get_min_tau(tau);
		if (isinf(tau[min_tau_sbi])) {
			printf("\n\n--------------- WARNING: min(tau) = +Inf - abort simulation ---------------\n\n");
			break;
		}

		h_current_time += tau[min_tau_sbi];
		gpuErrchk(hipMemcpy(d_current_time, &h_current_time, sizeof(float), hipMemcpyHostToDevice));

		// first we leap, with tau = min_tau, in every subvolume that has leap enabled
		leap_step<<<1, sbc>>>(d_state, d_reactants, d_products, d_rate_matrix, d_topology, d_react_rates_array,
				d_diff_rates_array, d_rrc, d_drc, tau[min_tau_sbi], d_current_time, d_leap, d_cr, d_prngstate);

		// now we do a single ssa step, if min was etc etc
		nsm_step<<<1, sbc>>>(d_state, d_reactants, d_products, d_topology, d_rate_matrix, d_rrc, d_drc,
				d_react_rates_array, d_diff_rates_array, thrust::raw_pointer_cast(tau.data()), min_tau_sbi,
				d_current_time, d_leap, d_prngstate);

		// update rates
		compute_rates<<<1, sbc>>>(d_state, d_reactants, d_topology, d_rate_matrix, d_rrc, d_drc, d_react_rates_array,
				d_diff_rates_array);

		// update tau array
		fill_tau_array_leap<<<1, sbc>>>(d_state, d_reactants, d_products, d_topology, d_rate_matrix,
				d_react_rates_array, d_diff_rates_array, thrust::raw_pointer_cast(tau.data()), d_leap, d_cr,
				d_prngstate);

#if LOG
		std::cout << "\n";
#endif

#if LOGSTEPS
		std::cout << "\n----- [step " << step << "] -----\n\n";

		// print system state
		gpuErrchk(hipMemcpy(h_state, d_state, sbc * spc * sizeof(int), hipMemcpyDeviceToHost));
		print_state(h_state, spc, sbc);

		// print rate matrix
		h_rate_matrix = new float[3 * sbc];
		gpuErrchk(hipMemcpy(h_rate_matrix, d_rate_matrix, 3 * sbc * sizeof(float), hipMemcpyDeviceToHost));
		print_rate_matrix(h_rate_matrix, sbc);

		// print tau array
		print_tau(tau, sbc);

		// print cr and leap arrays
		bool * h_leap = new bool[sbc];
		bool * h_cr = new bool[sbc];
		gpuErrchk(hipMemcpy(h_leap, d_leap, sbc * sizeof(bool), hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(h_cr, d_cr, sbc * sizeof(bool), hipMemcpyDeviceToHost));
		for (int i = 0; i < sbc; i++) {
			std::cout << "sbi " << i << "] " << "leap: " << (h_leap[i] ? "yes" : "no") << ", cr: "
					<< (h_cr[i] ? "yes" : "no") << "\n";
		}

#endif

	}

	gpuErrchk(hipDeviceSynchronize());

#if LOG
	std::cout << "\n--- End simulation.\n\n";
#endif

	gpuErrchk(hipMemcpy(h_state, d_state, sbc * spc * sizeof(int), hipMemcpyDeviceToHost));
	print_state(h_state, spc, sbc);
}

// ----- print utils stuff -----

void print_state(int * h_state, int spc, int sbc)
{
	std::cout << "\n--- [system state] ---\n";
	for (int i = 0; i < sbc; i++) {
		std::cout << "sbv " << i << ": ";
		for (int j = 0; j < spc; j++)
			std::cout << h_state[j * sbc + i] << " ";
		std::cout << "\n";
	}
	std::cout << "----------------------\n";
}

void print_rate_matrix(float * h_rate_matrix, int sbc)
{
	std::cout << "\n--- [rate matrix] ---\n";
	for (int i = 0; i < sbc; i++) {
		std::cout << "sbv " << i << ": ";
		std::cout << h_rate_matrix[i] << " ";
		std::cout << h_rate_matrix[i + sbc] << " ";
		std::cout << h_rate_matrix[i + sbc * 2] << "\n";
	}
	std::cout << "---------------------\n\n";
}

void print_tau(thrust::device_vector<float> tau, int sbc)
{
	std::cout << "\n--- [tau array] ---\n";
	for (int i = 0; i < sbc; i++)
		std::cout << "sbv " << i << ": " << tau[i] << "\n";
	std::cout << "-------------------\n\n";
}

}
