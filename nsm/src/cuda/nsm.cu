#include "hip/hip_runtime.h"
#include "../../include/cuda/nsm.cuh"

__global__ void fill_tau_array(float * tau, int sbc)
{
	int sbi = blockIdx.x * blockDim.x + threadIdx.x;
	if (sbi >= sbc)
		return;

	hiprandState s;
	hiprand_init(sbi, 0, 0, &s);

	float x = hiprand_uniform(&s);
	tau[sbi] = x;
	printf("tau[%d] = %f\n", sbi, x);
}

void h_fill_tau_array(thrust::device_vector<float> tau)
{
	fill_tau_array<<<1, tau.size()>>>(thrust::raw_pointer_cast(tau.data()), tau.size());
}
