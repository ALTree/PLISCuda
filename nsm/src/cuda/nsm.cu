#include "hip/hip_runtime.h"
#include "../../include/cuda/nsm.cuh"

// #define DEBUG

__device__ float react_rate(int * state, int * reactants, int sbc, int spc, int rc, int sbi, int ri, float * rrc)
{
#ifdef DEBUG
	printf("---------- begin react_rate( ) ---------- \n");
	printf("#reactions = %d, #species = %d, #subs = %d\n", rc, spc, sbc);
	printf("sub_index = %d, reaction_index = %d\n", sbi, ri);
#endif

	// search for the first specie in the reactions array that
	// does have a positive coefficent
	int index1 = ri;
	int specie_index = 0;
	while (reactants[index1] == 0) {
		index1 += rc;
		specie_index++;
	}

	if (reactants[index1] == 2) {    // bi_same reaction type
		// get specie count for that specie in the current subvolume
		// int specie_count = state[specie_index * subvolumes_count + subvolume_index];
		int specie_count = state[CUDA_GET_SPI(specie_index, sbi, sbc)];
#ifdef DEBUG
		printf("hit bi_same. Specie_index = %d, specie_count = %d\n", specie_index, species_count);
		printf("----------   end react_rate( ) ---------- \n\n");
#endif
		return 0.5 * specie_count * (specie_count - 1) * rrc[ri];
	}

	// if specie_index == # of species we are in a uni reaction
	if (specie_index != spc - 1) {

		// search for a possibile other specie with positive coefficient
		int index2 = index1 + rc;
		int specie_index2 = specie_index + 1;
		while (reactants[index2] == 0 && index2 < spc * rc) {
			index2 += rc;
			specie_index2++;
		}

		if (reactants[index2] != 0) {    // bi_diff reaction type
			int specie1_count = state[CUDA_GET_SPI(specie_index, sbi, sbc)];
			int specie2_count = state[CUDA_GET_SPI(specie_index2, sbi, sbc)];
#ifdef DEBUG
			printf("hit bi_diff. Specie_index1 = %d, specie_index2 = %d,\n", specie_index, specie_index2);
			printf("    specie1_count = %d, specie2_count = %d\n", specie1_count, specie2_count);
			printf("----------   end react_rate( ) ----------\n\n");
#endif
			return specie1_count * specie2_count * rrc[ri];
		}
	}

	// uni reaction type
	int specie_count = state[CUDA_GET_SPI(specie_index, sbi, sbc)];
#ifdef DEBUG
	printf("hit uni. Specie_index = %d, specie_count = %d, ", specie_index, specie_count);
	printf("----------   end react_rate( ) ---------- \n\n");
#endif
	return specie_count * rrc[ri];
}

__device__ float * react_rates(int * state, int * reactants, int sbc, int spc, int rc, int sbi, float * rrc)
{
	__shared__ extern float react_rates_array[];    // we need extern because the size is not a compile-time constant
													// we'll need to allocate during the kernel invocation
													// TODO: rethink about this

	for (int i = 0; i < rc; i++) {
		react_rates_array[i] = react_rate(state, reactants, sbc, spc, rc, sbi, i, rrc);
	}

	return react_rates_array;
}

__device__ float * diff_rates(int * state, int subvolumes_count, int species_count, int subvolume_index,
		float * diffusion_rates_constants)
{
	__shared__ extern float diffusion_rates_array[];
	for (int i = 0; i < species_count; i++) {
		diffusion_rates_array[i] = diffusion_rates_constants[i]
				* state[CUDA_GET_SPI(i, subvolume_index, subvolumes_count)];
	}

	return diffusion_rates_array;
}

/*
__device__ void rate_matrix_row(int * state, int * reactants, int subvolumes_count, int species_count,
		int reactions_count, float * reaction_rate_constants, float * diffusion_rate_constants, float * rate_matrix,
		int subvolume_index)
{
	// compute new reaction rates
	float * react_rates_array = react_rates(reactants, reactions_count, state, subvolumes_count, species_count,
			subvolume_index, reaction_rate_constants);
	float reactions_rates_sum = sum_fp_array(react_rates_array, reactions_count);

	// compute new diffusion rates
	float * diff_rates_array = diff_rates(state, subvolumes_count, species_count, subvolume_index,
			diffusion_rate_constants);
	float diffusion_rates_sum = sum_fp_array(diff_rates_array, species_count);

	// update rate matrix
	rate_matrix[subvolume_index] = reactions_rates_sum;
	rate_matrix[subvolume_index * 2] = diffusion_rates_sum;
	rate_matrix[subvolume_index * 3] = reactions_rates_sum + diffusion_rates_sum;
}
*/
