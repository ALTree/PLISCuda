#include "hip/hip_runtime.h"
#include "../../include/cuda/nsm.cuh"

__device__ int choose_rand_reaction(int sbc, int rc, float * rate_matrix, float * react_rates_array, float rand)
{
	int sbi = blockIdx.x * blockDim.x + threadIdx.x;
	if (sbi >= sbc)
		return -1;

	float sum = rate_matrix[sbi];
	float scaled_sum = sum * rand;
	float partial_sum = 0;

	int ri = 0;
	while(partial_sum <= scaled_sum) {
		partial_sum += react_rates_array[ri * sbc + sbi];
		ri++;
	}

	return ri-1;
}

__global__ void fill_tau_array(float * tau, int sbc)
{
	int sbi = blockIdx.x * blockDim.x + threadIdx.x;
	if (sbi >= sbc)
		return;

	hiprandState s;
	// hiprandStateMRG32k3a s;
	hiprand_init(sbi, 0, 0, &s);

	float x = hiprand_uniform(&s);
	tau[sbi] = x;
}

void h_fill_tau_array(thrust::device_vector<float> &tau)
{
	fill_tau_array<<<1, tau.size()>>>(thrust::raw_pointer_cast(tau.data()), tau.size());
}

int h_get_min_tau(thrust::device_vector<float> &tau)
{
	thrust::device_vector<float>::iterator iter = thrust::min_element(tau.begin(), tau.end());
	return iter - tau.begin();
}
