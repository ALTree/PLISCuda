#include "hip/hip_runtime.h"
#include "../../include/cuda/nsm.cuh"

#define DEBUG

__device__ int choose_rand_reaction(int rc, float * rate_matrix, float * react_rates_array, float rand)
{
	int sbi = blockIdx.x * blockDim.x + threadIdx.x;
	if (sbi >= SBC)
		return -1;

	// if R (the sum of the reaction rates) is zero,
	// we can't fire any reaction
	if (rate_matrix[GET_RATE(0, sbi)] == 0)
		return -1;

	float sum = rate_matrix[sbi];
	float scaled_sum = sum * rand;
	float partial_sum = 0;

	int ri = 0;
	while (partial_sum <= scaled_sum) {
		partial_sum += react_rates_array[ri * SBC + sbi];
		ri++;
	}

	return ri - 1;
}

__device__ int choose_rand_specie(int * topology, float * rate_matrix, float * diff_rates_array,
		float rand)
{
	int sbi = blockIdx.x * blockDim.x + threadIdx.x;
	if (sbi >= SBC)
		return -1;

	// if D (the sum of the diffusion rates) is zero,
	// we can't diffuse any specie
	if (rate_matrix[GET_RATE(1, sbi)] == 0)
		return -1;

	int neigh_count = 0;
	for (int i = 0; i < 6; i++)
		neigh_count += (topology[sbi * 6 + i] != -1);

	// we need to scale back rate_matrix[2][sbi] before performing
	// the linear scaling

	float sum = rate_matrix[SBC * 1 + sbi] / neigh_count;
	float scaled_sum = sum * rand;
	float partial_sum = 0;

	int spi = 0;
	while (partial_sum <= scaled_sum) {
		partial_sum += diff_rates_array[spi * SBC + sbi];
		spi++;
	}

	return spi - 1;
}

__global__ void fill_tau_array(float * tau, float * rate_matrix)
{
	int sbi = blockIdx.x * blockDim.x + threadIdx.x;
	if (sbi >= SBC)
		return;

	hiprandStateMRG32k3a s;
	hiprand_init(2 * sbi, 0, 0, &s);    // initialize with *2sbi to avoid getting the same first value
									   // later when we use hiprand_init(sbi, ..)

	float rand = hiprand_uniform(&s);
	tau[sbi] = -logf(rand) / rate_matrix[GET_RATE(2, sbi)];
}

int h_get_min_tau(thrust::device_vector<float> &tau)
{
	thrust::device_vector<float>::iterator iter = thrust::min_element(tau.begin(), tau.end());
	return iter - tau.begin();
}

__global__ void nsm_step(int * state, int * reactants, int * products, int * topology, int rc,
		float * rate_matrix, float * rrc, float * drc, float * react_rates_array, float * diff_rates_array, float * tau,
		int min_sbi, int step)
{

	int sbi = blockIdx.x * blockDim.x + threadIdx.x;
	if (sbi >= SBC)
		return;

	// create and initialize thread's prng
	hiprandStateMRG32k3a s;
	hiprand_init(sbi, 0, step, &s);

	float rand = hiprand_uniform(&s);

#ifdef DEBUG
	printf("[sbv %d] tau = %f, rand = %f\n", sbi, tau[sbi], rand);
#endif

	if (rand < rate_matrix[GET_RATE(0, sbi)] / rate_matrix[GET_RATE(2, sbi)]) {
		// fire a reaction

		// choose a random reaction to fire
		int ri = choose_rand_reaction(rc, rate_matrix, react_rates_array, rand);

		if (ri == -1)    // we can't fire any reaction in this subvolume
			goto UPDATE_TAU;

#ifdef DEBUG
		if (ri >= rc) {
			printf(">>>>>>>>>>>>>>>> ARGH! @ [subv %d]: random reaction index = %d\n", sbi, ri);
		}
#endif

		if (sbi == min_sbi) {
			printf("(%f) [subv %d] fire reaction %d\n", tau[sbi], sbi, ri);
		}

		// fire reaction and update the state of the system
		if (sbi == min_sbi) {    // (but only if you are the choosen one)
			for (int i = 0; i < SPC; i++)
				state[GET_SPI(i, sbi)] += products[i * rc + ri] - reactants[i * rc + ri];
		}

		// TODO: do we need this?
		__syncthreads();

		// update rate matrix
		react_rates(state, reactants, rc, rrc, react_rates_array);
		diff_rates(state, drc, diff_rates_array);
		update_rate_matrix(topology, rc, rate_matrix, react_rates_array, diff_rates_array);
	} else {
		// diffuse a specie

		// choose a random specie to diffuse
		int spi = choose_rand_specie(topology, rate_matrix, diff_rates_array, rand);

#ifdef DEBUG
		if (spi >= SPC) {
			printf(">>>>>>>>>>>>>>>> ARGH! @ [subv %d] random specie index = %d\n", sbi, spi);
		}
#endif

		// choose a random destination
		// TODO: we need to re-use the rand we already have.
		//       Also find a better way to ensure fairness on
		//       index 5.
		int rdi = (int) (hiprand_uniform(&s) * 6);
		while (rdi > 5)
			rdi = (int) hiprand_uniform(&s);

		// get index of neighbour #rdi (overwrite rdi, whatever)
		rdi = topology[sbi * 6 + rdi];

#ifdef DEBUG
		if (rdi >= SBC) {
			printf(">>>>>>>>>>>>>>>> ARGH! @ [subv %d] random neigh = %d\n", sbi, rdi);
		}
#endif

		if (sbi == min_sbi) {
			printf("(%f) [subv %d] diffuse specie %d in subvolume %d\n", tau[sbi], sbi, spi, rdi);
		}

		// Update state iff we are the choosen one.
		// Also if we hit a -1 (i.e. diffuse to myself) don't do anything
		if (sbi == min_sbi && rdi != -1) {
			state[GET_SPI(spi, sbi)] -= 1;
			state[GET_SPI(spi, rdi)] += 1;
		}

		// TODO: do we need this?
		__syncthreads();

		// update rate matrix
		// The destination subvolume will update its own rates... right?
		react_rates(state, reactants,rc, rrc, react_rates_array);
		diff_rates(state, drc, diff_rates_array);
		update_rate_matrix(topology, rc, rate_matrix, react_rates_array, diff_rates_array);
	}

	// compute next event time for this subvolume
	UPDATE_TAU: rand = hiprand_uniform(&s);
	tau[sbi] = -logf(rand) / rate_matrix[GET_RATE(2, sbi)] + tau[min_sbi];
}
