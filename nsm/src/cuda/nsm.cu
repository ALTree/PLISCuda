#include "hip/hip_runtime.h"
#include "../../include/cuda/nsm.cuh"

// #define DEBUG

__device__ float react_rate(int * reactants, int reactions_count, int * state, int species_count, int subvolumes_count,
		int subvolume_index, float * reaction_rate_constants, int reaction_number)
{
#ifdef DEBUG
	printf("---------- begin react_rate( ) ---------- \n");
	printf("#reactions = %d, #species = %d, #subs = %d\n", reactions_count, species_count, subvolumes_count);
	printf("sub_index = %d, reaction_number = %d\n", subvolume_index, reaction_number);
#endif

	// search for the first specie in the reactions array that
	// does have a positive coefficent
	int index1 = reaction_number;
	int specie_index = 0;
	while (reactants[index1] == 0) {
		index1 += reactions_count;
		specie_index++;
	}

	if (reactants[index1] == 2) {    // bi_same reaction type
		// get specie count for that specie in the current subvolume
		// int specie_count = state[specie_index * subvolumes_count + subvolume_index];
		int specie_count = state[CUDA_GET_SPI(specie_index, subvolume_index, subvolumes_count)];
#ifdef DEBUG
		printf("hit bi_same. Specie_index = %d, specie_count = %d\n", specie_index, species_count);
		printf("----------   end react_rate( ) ---------- \n\n");
#endif
		return 0.5 * specie_count * (specie_count - 1) * reaction_rate_constants[reaction_number];
	}

	// if specie_index == # of species we are in a uni reaction
	if (specie_index != species_count - 1) {

		// search for a possibile other specie with positive coefficient
		int index2 = index1 + reactions_count;
		int specie_index2 = specie_index + 1;
		while (reactants[index2] == 0 && index2 < species_count * reactions_count) {
			index2 += reactions_count;
			specie_index2++;
		}

		if (reactants[index2] != 0) {    // bi_diff reaction type
			int specie1_count = state[CUDA_GET_SPI(specie_index, subvolume_index, subvolumes_count)];
			int specie2_count = state[CUDA_GET_SPI(specie_index2, subvolume_index, subvolumes_count)];
#ifdef DEBUG
			printf("hit bi_diff. Specie_index1 = %d, specie_index2 = %d,\n", specie_index, specie_index2);
			printf("    specie1_count = %d, specie2_count = %d\n", specie1_count, specie2_count);
			printf("----------   end react_rate( ) ----------\n\n");
#endif
			return specie1_count * specie2_count * reaction_rate_constants[reaction_number];
		}
	}

	// uni reaction type
	int specie_count = state[CUDA_GET_SPI(specie_index, subvolume_index, subvolumes_count)];
#ifdef DEBUG
	printf("hit uni. Specie_index = %d, specie_count = %d, ", specie_index, specie_count);
	printf("----------   end react_rate( ) ---------- \n\n");
#endif
	return specie_count * reaction_rate_constants[reaction_number];
}

__device__ float * react_rates(int * reactants, int reactions_count, int * state, int subvolumes_count,
		int species_count, int subvolume_index, float * reaction_rate_constants)
{
	__shared__ extern float react_rates_array[];    // we need extern because the size is not a compile-time constant
													// we'll need to allocate during the kernel invocation
													// TODO: rethink about this

	for (int i = 0; i < reactions_count; i++) {
		react_rates_array[i] = react_rate(reactants, reactions_count, state, species_count, subvolumes_count,
				subvolume_index, reaction_rate_constants, i);
	}

	return react_rates_array;
}

__device__ float * diff_rates(int * state, int subvolumes_count, int species_count, int subvolume_index,
		float * diffusion_rates_constants)
{
	__shared__ extern float diffusion_rates_array[];
	for (int i = 0; i < species_count; i++) {
		diffusion_rates_array[i] = diffusion_rates_constants[i]
				* state[CUDA_GET_SPI(i, subvolume_index, subvolumes_count)];
	}

	return diffusion_rates_array;
}

__device__ void rate_matrix_row(int * state, int * reactants, int subvolumes_count, int species_count,
		int reactions_count, float * reaction_rate_constants, float * diffusion_rate_constants, float * rate_matrix,
		int subvolume_index)
{
	// compute new reaction rates
	float * react_rates_array = react_rates(reactants, reactions_count, state, subvolumes_count, species_count,
			subvolume_index, reaction_rate_constants);
	float reactions_rates_sum = sum_fp_array(react_rates_array, reactions_count);

	// compute new diffusion rates
	float * diff_rates_array = diff_rates(state, subvolumes_count, species_count, subvolume_index,
			diffusion_rate_constants);
	float diffusion_rates_sum = sum_fp_array(diff_rates_array, species_count);

	// update rate matrix
	rate_matrix[subvolume_index] = reactions_rates_sum;
	rate_matrix[subvolume_index * 2] = diffusion_rates_sum;
	rate_matrix[subvolume_index * 3] = reactions_rates_sum + diffusion_rates_sum;
}


