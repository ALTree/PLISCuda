#include "hip/hip_runtime.h"
#include "../../include/cuda/nsm.cuh"

__global__ void fill_tau_array(float * tau, int sbc)
{
	int sbi = blockIdx.x * blockDim.x + threadIdx.x;
	if (sbi >= sbc)
		return;

	hiprandState s;
	hiprand_init(sbi, 0, 0, &s);

	float x = hiprand_uniform(&s);
	tau[sbi] = x;
}

void h_fill_tau_array(thrust::device_vector<float> &tau)
{
	fill_tau_array<<<1, tau.size()>>>(thrust::raw_pointer_cast(tau.data()), tau.size());
}

int h_get_min_tau(thrust::device_vector<float> &tau)
{
	thrust::device_vector<float>::iterator iter = thrust::min_element(tau.begin(), tau.end());
	return iter - tau.begin();
}
