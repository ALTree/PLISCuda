#include "hip/hip_runtime.h"
#include "../include/nsm.cuh"

__device__ float react_rate(int * reactants, int reactions_count, int * state, int species_count, int subvolumes_count,
		int subvolume_index, double * reaction_rate_constants, int reaction_number)
{
	// search for the first specie in the reactions array that
	// does have a positive coefficent
	int index1 = reaction_number;
	int specie_index = 0;
	while (reactants[index1] == 0) {
		index1 += reactions_count;
		specie_index++;
	}

	if (reactants[index1] == 2) {    // bi_same reaction type
		// get specie count for that specie in the current subvolume
		int specie_count = state[specie_index * subvolumes_count + subvolume_index];
		return 0.5 * specie_count * (specie_count - 1) * reaction_rate_constants[reaction_number];
	}

	// search for a possibile other specie with positive coefficient
	int index2 = index1 + reactions_count;
	int specie_index2 = specie_index + 1;
	while (reactants[index2] == 0 && index2 < species_count * reactions_count) {
		index2 += reactions_count;
		specie_index2++;
	}

	if (reactants[index2] != 0) {    // bi_diff reaction type
		int specie1_count = state[specie_index * subvolumes_count + subvolume_index];
		int specie2_count = state[specie_index2 * subvolumes_count + subvolume_index];
		return specie1_count * specie2_count * reaction_rate_constants[reaction_number];
	}

	// uni reaction type
	int specie_count = state[specie_index * subvolumes_count + subvolume_index];
	return specie_count * reaction_rate_constants[reaction_number];
}

__device__ float * react_rates(int * reactants, int reactions_count, int * state, int subvolumes_count,
		int species_count, int subvolume_index, double * reaction_rate_constants)
{
	__shared__ extern float react_rates_array[];    // we need extern because the size is not a compile-time constant
													// we'll need to allocate during the kernel invocation

	for (int i = 0; i < reactions_count; i++) {
		react_rates_array[i] = react_rate(reactants, reactions_count, state, species_count, subvolumes_count,
				subvolume_index, reaction_rate_constants, i);
	}

	return react_rates_array;
}

__global__ void test()
{
	printf("hello!\n");
}

void foo()
{
	test<<<10, 1>>>();
}
