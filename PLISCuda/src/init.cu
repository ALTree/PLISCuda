#include "hip/hip_runtime.h"
#include "../include/cuda/init.cuh"

__global__ void init_prng(hiprandStateMRG32k3a * prngstate)
{
	INDCHECK()

#ifndef TEST
	hiprand_init(clock64() * sbi, 0, 0, &prngstate[sbi]);
#else // fixed seeds for a deterministic simulation
	hiprand_init(sbi, 0, 0, &prngstate[sbi]);
#endif

}

__global__ void init_ncount(neigh neigh)
{
	INDCHECK()

	int nc = 0;
	for (int i = 0; i < 6; i++)
		nc += (neigh.index[sbi * 6 + i] != sbi);
	
	neigh.count[sbi] = nc;

}

__device__ int HOR(reactions reactions, int spi)
{
	int max_hor = 0;
	bool is_bi_reaction = false;

	for (int ri = 0; ri < RC; ri++) {
		// if spi is not a reactant of the current reaction, continue
		// with the next one.
		if (reactions.r[GET_COEFF(spi, ri)] == 0)
			continue;

		// sum all the coeff. of the current reaction to compute its
		// order.
		int hor = 0;
		for (int j = 0; j < SPC; j++) {
			int c = reactions.r[GET_COEFF(j, ri)];
			hor += c;
			// check if ri requires 2 molecules of spi
			if (j == spi && c == 2) {
				is_bi_reaction = true;    // TODO: replace with branchless code
			}
		}

		max_hor = max(hor, max_hor);
	}

	if (is_bi_reaction)
		max_hor = 3;

	return max_hor;
}


__global__ void init_hors(int * hors, reactions reactions, int spc)
{
	unsigned int sbi = blockIdx.x * blockDim.x + threadIdx.x;
	if (sbi != 0)
		return;

	for (int spi = 0; spi < spc; spi++)
		hors[spi] = HOR(reactions, spi);
}


